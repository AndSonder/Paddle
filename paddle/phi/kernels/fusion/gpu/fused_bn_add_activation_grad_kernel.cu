#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_dnn.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/activation_functor.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/norm_utils.h"
#include "paddle/phi/kernels/fused_bn_add_activation_grad_kernel.h"

PHI_DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace phi {
namespace fusion {

template <typename T>
using CudnnDataType = phi::backends::gpu::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T, typename Context>
void FusedBatchNormAddActGradKernel(const Context &dev_ctx,
                                    const DenseTensor &x,
                                    const DenseTensor &y,
                                    const DenseTensor &y_grad,
                                    const DenseTensor &scale,
                                    const DenseTensor &bias,
                                    const DenseTensor &saved_mean,
                                    const DenseTensor &saved_variance,
                                    const DenseTensor &reserve_space,
                                    float momentum,
                                    float epsilon,
                                    const std::string &act_type,
                                    DenseTensor *x_grad,
                                    DenseTensor *z_grad,
                                    DenseTensor *scale_grad,
                                    DenseTensor *bias_grad) {
#if CUDNN_VERSION < 7401
  PADDLE_THROW(phi::errors::Unimplemented(
      "The fused_bn_add_activation operator is not supported on GPU "
      "when CUDNN version < 7.4.1"));
#endif
  bool is_gpu_place = dev_ctx.GetPlace().GetType() == phi::AllocationType::GPU;
  PADDLE_ENFORCE_EQ(is_gpu_place,
                    true,
                    phi::errors::PreconditionNotMet("It must use CUDAPlace."));

  const auto *d_y = &y;
  const auto &in_dims = x.dims();

  int N, C, H, W, D;
  const DataLayout data_layout = DataLayout::kNHWC;
  phi::funcs::ExtractNCWHD(in_dims, data_layout, &N, &C, &H, &W, &D);

  dev_ctx.template Alloc<T>(x_grad);
  dev_ctx.template Alloc<T>(z_grad);
  PADDLE_ENFORCE_EQ(
      scale_grad && bias_grad,
      true,
      phi::errors::PreconditionNotMet(
          "Both the scale grad and the bias grad must not be null."));
  dev_ctx.template Alloc<BatchNormParamType<T>>(scale_grad);
  dev_ctx.template Alloc<BatchNormParamType<T>>(bias_grad);
  PADDLE_ENFORCE_EQ(
      scale.dims().size(),
      1UL,
      phi::errors::PreconditionNotMet("The scale only has one dimension."));
  PADDLE_ENFORCE_EQ(
      scale.dims()[0],
      C,
      phi::errors::PreconditionNotMet(
          "The size of scale is equal to the channel of Input(X)."));

  std::vector<int> dims = {N, C, H, W, D};
  std::vector<int> strides = {H * W * C * D, 1, W * D * C, D * C, C};
  // ------------------- cudnn descriptors ---------------------
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t bn_param_desc_;
  hipdnnBatchNormMode_t mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;

  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(static_cast<double>(epsilon), HIPDNN_BN_MIN_EPSILON);

  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipdnnSetTensorNdDescriptor(
      data_desc_,
      CudnnDataType<T>::type,
      in_dims.size() > 3 ? in_dims.size() : 4,
      dims.data(),
      strides.data()));
  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipdnnDeriveBNTensorDescriptor(
      bn_param_desc_, data_desc_, mode_));

  const auto *saved_mean_data =
      saved_mean.template data<BatchNormParamType<T>>();
  const auto *saved_var_data =
      saved_variance.template data<BatchNormParamType<T>>();

  size_t workspace_size = 0;
  void *workspace_ptr = nullptr;
  phi::DenseTensor workspace_tensor;
  auto reserve_space_size = reserve_space.memory_size();
  cudnnBatchNormOps_t bnOps_ = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
  phi::backends::gpu::ScopedActivationDescriptor scope_act_desc;
  hipdnnActivationDescriptor_t activation_desc_ =
      scope_act_desc.descriptor<T>(act_type);
  // --------------- cudnn batchnorm workspace ---------------
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::cudnnGetBatchNormalizationBackwardExWorkspaceSize(
          /*handle=*/dev_ctx.cudnn_handle(),
          /*mode=*/mode_,
          /*bnOps=*/bnOps_,
          /*xDesc=*/data_desc_,
          /*yDesc=*/data_desc_,
          /*dyDesc=*/data_desc_,
          /*dzDesc=*/data_desc_,
          /*dxDesc=*/data_desc_,
          /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
          /*activationDesc=*/activation_desc_,
          /*sizeInBytes=*/&workspace_size));

  workspace_tensor.Resize({static_cast<int64_t>(workspace_size)});
  workspace_ptr = dev_ctx.template Alloc<T>(&workspace_tensor);

  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::cudnnBatchNormalizationBackwardEx(
      /*handle=*/dev_ctx.cudnn_handle(),
      /*mode=*/mode_,
      /*bnOps=*/bnOps_,
      /*alphaDataDiff=*/CudnnDataType<T>::kOne(),
      /*betaDataDiff=*/CudnnDataType<T>::kZero(),
      /*alphaParamDiff=*/CudnnDataType<T>::kOne(),
      /*betaParamDiff=*/CudnnDataType<T>::kZero(),
      /*xDesc=*/data_desc_,
      /*xData=*/x.template data<T>(),
      /*yDesc=*/data_desc_,
      /*yData=*/y.template data<T>(),
      /*dyDesc=*/data_desc_,
      /*dyData=*/d_y->template data<T>(),
      /*dzDesc=*/data_desc_,
      /*dzData=*/z_grad->template data<T>(),
      /*dxDesc=*/data_desc_,
      /*dxData=*/x_grad->template data<T>(),
      /*dBnScaleBiasDesc=*/bn_param_desc_,
      /*bnScaleData=*/scale.template data<BatchNormParamType<T>>(),
      /*bnBiasData=*/bias.template data<BatchNormParamType<T>>(),
      /*dBnScaleData=*/scale_grad->template data<BatchNormParamType<T>>(),
      /*dBnBiasData=*/bias_grad->template data<BatchNormParamType<T>>(),
      /*epsilon=*/epsilon,
      /*savedMean=*/saved_mean_data,
      /*savedInvVariance=*/saved_var_data,
      /*activationDesmc=*/activation_desc_,
      /*workspace=*/workspace_ptr,
      /*workSpaceSizeInBytes=*/workspace_size,
      /*reserveSpace=*/const_cast<T *>(reserve_space.template data<T>()),
      /*reserveSpaceSizeInBytes=*/reserve_space_size));

  // clean when exit.
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_bn_add_activation_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedBatchNormAddActGradKernel,
                   phi::dtype::float16) {
  kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
  kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
}
